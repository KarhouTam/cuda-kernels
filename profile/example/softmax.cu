#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <torch/types.h>

template <typename T>
__device__ T warpReduceSum(T val) {
#pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        val += __shfl_xor_sync(0xFFFFFFFF, val, offset);
    }
    return val;
}

template <typename T>
__device__ T warpReduceMax(T val) {
#pragma unroll
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
        val = fmaxf(val, __shfl_xor_sync(0xFFFFFFFF, val, offset));
    }
    return val;
}

__global__ void softmax_kernel(float* input, float* output, const int M, const int N) {
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    const int numWarps = warpsPerBlock * gridDim.x;
    const int idx = warpsPerBlock * blockIdx.x + warpId;
    for (int m = idx; m < M; m += numWarps) {
        // each lane (thread in a warp) calculate the maxval among
        // data with indices [landId, landId + 32, laneId + 64, ...]
        const float* x = input + m * N;
        float* const y = output + m * N;

        float maxval = -INFINITY;
        for (int i = laneId; i < N; i += warpSize) {
            maxval = fmaxf(maxval, x[i]);
        }
        // warp-reduce to calculate the MAX of maxval among all lanes
        // and the 0-th lane will store the result
        maxval = warpReduceMax(maxval);

        float sum = 0.0f;
        for (int i = laneId; i < N; i += warpSize) {
            sum += expf(x[i] - maxval);
        }

        sum = warpReduceSum(sum);
        for (int i = laneId; i < N; i += warpSize) {
            y[i] = expf(x[i] - maxval) / sum;
        }
    }
}
torch::Tensor softmax(torch::Tensor input) {
    const int blockSize = 128;
    const auto M = input.size(0);
    const auto N = input.size(1);

    auto result = torch::empty_like(input);

    dim3 blockDim(blockSize);
    dim3 gridDim(M * 32 / blockDim.x);

    softmax_kernel<<<gridDim, blockDim>>>(input.data_ptr<float>(), result.data_ptr<float>(), M, N);
    return result;
}
