#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>

#include "common.h"

/* Softmax forward implementation

Usage: ./softmax_forward <kernel> [blockSize]
e.g. ./softmax_forward 1

softmax_forward_cpu(): CPU implementation

softmax_forward_kernel1(): Naive implementation on CUDA. Each thread handles
one row of the input.

softmax_forward_kernel2(): Optimized implementation on CUDA. Compares to
kernel1, each warp (32 threads) handles one row.

online_softmax_forward_kernel3(): Online softmax forward implementation on CUDA.
Also each warp handles one row of the input.

(Fastest)
softmax_forward_kernel4(): Online softmax forward implementation on CUDA.
Each warp handles one row of the input.
Use float4 to acclerate memory access.

*/

void softmax_cpu(float* input, float* output, const int M, const int N) {
    for (int m = 0; m < M; ++m) {
        float maxval = -INFINITY;
        const float* x = input + m * N;
        for (int n = 0; n < N; ++n) {
            maxval = maxval > x[n] ? maxval : x[n];
        }
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            s += exp(x[n] - maxval);
        }
        float* y = output + m * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

void online_softmax_cpu(float* input, float* output, const int M, const int N) {
    for (int m = 0; m < M; ++m) {
        const float* x = input + m * N;
        float maxval = -INFINITY;
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            if (maxval < x[n]) {
                s *= exp(maxval - x[n]);
                maxval = x[n];
            }
            s += exp(x[n] - maxval);
        }

        float* y = output + m * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

__global__ void softmax_kernel1(float* input, float* output, const int M,
                                const int N) {
    // naive implementation
    // one thread one row
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int idx = bid * blockDim.x + tid;
    if (idx < M) {
        float maxval = -INFINITY;
        const float* x = input + idx * N;
        for (int n = 0; n < N; ++n) {
            maxval = maxval > x[n] ? maxval : x[n];
        }
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            s += exp(x[n] - maxval);
        }
        float* const y = output + idx * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

__global__ void softmax_kernel2(float* input, float* output, const int M,
                                const int N) {
    // use more threads per row than kernel1
    // one warp (32 threads) process one row
    // use warp reduce functions
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    const int numWarps = warpsPerBlock * gridDim.x;
    const int idx = warpsPerBlock * blockIdx.x + warpId;
    for (int m = idx; m < M; m += numWarps) {
        // each lane (thread in a warp) calculate the maxval among
        // data with indices [landId, landId + 32, laneId + 64, ...]
        const float* x = input + m * N;
        float* const y = output + m * N;

        float maxval = -INFINITY;
        for (int i = laneId; i < N; i += warpSize) {
            maxval = fmaxf(maxval, x[i]);
        }
        // warp-reduce to calculate the MAX of maxval among all lanes
        // and the 0-th lane will store the result
        maxval = warpReduceMax(maxval);

        float sum = 0.0f;
        for (int i = laneId; i < N; i += warpSize) {
            sum += expf(x[i] - maxval);
        }

        sum = warpReduceSum(sum);
        for (int i = laneId; i < N; i += warpSize) {
            y[i] = expf(x[i] - maxval) / sum;
        }
    }
}

__global__ void online_softmax_kernel3(float* input, float* output, const int M,
                                       const int N) {
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    const int numWarps = warpsPerBlock * gridDim.x;
    const int idx = warpsPerBlock * blockIdx.x + warpId;
    for (int m = idx; m < M; m += numWarps) {
        const float* x = input + m * N;
        float* const y = output + m * N;
        float maxval = -INFINITY, sum = 0.0f, bigger;
        for (int i = laneId; i < N; i += warpSize) {
            bigger = fmaxf(maxval, x[i]);
            sum = sum * expf(maxval - bigger) + expf(x[i] - bigger);
        }

        float offsetMax, offsetSum;
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            __syncwarp();
            offsetMax = __shfl_xor_sync(0xFFFFFFFF, maxval, offset);
            offsetSum = __shfl_xor_sync(0xFFFFFFFF, sum, offset);
            if (offsetMax > maxval) {
                sum *= expf(maxval - offsetMax);
                maxval = offsetMax;
            } else {
                offsetSum *= expf(offsetMax - maxval);
            }
            sum += offsetSum;
        }
        for (int i = laneId; i < N; i += warpSize) {
            y[i] = expf(x[i] - maxval) / sum;
        }
    }
}

__global__ void online_softmax_kernel4(float* __restrict__ input,
                                       float* __restrict__ output, const int M,
                                       const int N) {
    // this kernel is f*cking faster than any other kernels!
    // use float4 to acclerate memory access
    // each warp (32 threads) handles one row
    using f128 = Package128<float>;
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    int row = warpsPerBlock * blockIdx.x + warpId;
    if (row < M) {
        float* x = input + row * N;
        float* y = output + row * N;
        float laneMax = -INFINITY, laneSum = 0.0f;
        for (int i = laneId * f128::size; i < N; i += warpSize * f128::size) {
            f128 xi = load128cs(x + i);
            float packMax = -INFINITY, packSum = 0.0f;
            #pragma unroll
            for (int k = 0; k < f128::size; ++k) {
                float newPackMax = fmaxf(packMax, xi[k]);
                packSum = expf(packMax - newPackMax) * packSum +
                          expf(xi[k] - newPackMax);
                packMax = newPackMax;
            }
            float newLaneMax = fmaxf(laneMax, packMax);
            laneSum = laneSum * expf(laneMax - newLaneMax) +
                      packSum * expf(packMax - newLaneMax);
            laneMax = newLaneMax;
        }

        float maxVal = laneMax, sumVal = laneSum;
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            float offsetMax = __shfl_xor_sync(0xFFFFFFFF, maxVal, offset);
            float offsetSum = __shfl_xor_sync(0xFFFFFFFF, sumVal, offset);
            if (maxVal > offsetMax) {
                sumVal += expf(offsetMax - maxVal) * offsetSum;
            } else {
                sumVal = sumVal * expf(maxVal - offsetMax) + offsetSum;
                maxVal = offsetMax;
            }
        }
        for (int i = laneId * f128::size; i < N; i += warpSize * f128::size) {
            f128 out;
            f128 xi = load128cs(x + i);
            #pragma unroll
            for (int k = 0; k < f128::size; ++k) {
                out[k] = expf(xi[k] - maxVal) / sumVal;
            }
            store128cs(y + i, out);
        }
    }
}
#define M 8196
#define N 8196
#define BLOCK_SIZE 128

int main(int argc, char** argv) {
    if (argc < 2) {
        fprintf(stderr, "Usage: softmax_forward <kernel> [blockSize]\n");
        return EXIT_FAILURE;
    }
    int kernel = atoi(argv[1]);

    int blockSize = BLOCK_SIZE;
    if (argc > 2) {
        blockSize = atoi(argv[2]);
    }

    float* input = (float*)malloc(M * N * sizeof(float));
    float* output = (float*)malloc(M * N * sizeof(float));
    float* resFromGPU = (float*)malloc(M * N * sizeof(float));
    initArrFloat(input, M * N);

    float *inputGPU, *outputGPU;
    cudaErrorCheck(hipMalloc(&inputGPU, M * N * sizeof(float)));
    cudaErrorCheck(hipMemcpy(inputGPU, input, M * N * sizeof(float),
                              hipMemcpyHostToDevice));
    cudaErrorCheck(hipMalloc(&outputGPU, M * N * sizeof(float)));

    online_softmax_cpu(input, output, M, N);

    switch (kernel) {
        case 1:
            softmax_kernel1<<<M * N / blockSize, blockSize>>>(inputGPU,
                                                              outputGPU, M, N);
            break;
        case 2:
            softmax_kernel2<<<ceilDiv(M * 32, blockSize), blockSize>>>(
                inputGPU, outputGPU, M, N);
            break;

        case 3:
            online_softmax_kernel3<<<ceilDiv(M * 32, blockSize), blockSize>>>(
                inputGPU, outputGPU, M, N);
            break;

        case 4:
            online_softmax_kernel4<<<ceilDiv(M * 32, blockSize), blockSize,
                                     0>>>(inputGPU, outputGPU, M, N);
            break;

        default:
            printf("Error: Invalid kernel type: %i\n", kernel);
            return EXIT_FAILURE;
    }
    cudaErrorCheck(hipDeviceSynchronize());
    cudaErrorCheck(hipMemcpy(resFromGPU, outputGPU, M * N * sizeof(float),
                              hipMemcpyDeviceToHost));

    float elapsedTime;
    if (checkResults(output, resFromGPU, M * N)) {
        switch (kernel) {
            case 1:
                benchmarkKernel(softmax_kernel1, M * N / blockSize, blockSize,
                                0, 0, &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 2:
                benchmarkKernel(softmax_kernel2, ceilDiv(M * 32, blockSize),
                                blockSize, 0, 0, &elapsedTime, inputGPU,
                                outputGPU, M, N);
                break;
            case 3:
                benchmarkKernel(online_softmax_kernel3,
                                ceilDiv(M * 32, blockSize), blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 4:
                benchmarkKernel(online_softmax_kernel4,
                                ceilDiv(M * 32, blockSize), blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
        }
        printf(
            "softmax_forward kernel: %i | matrixSize: %i x %i | Times: %f ms | "
            "blockSize: %i\n",
            kernel, M, N, elapsedTime, blockSize);
    }

    free(input);
    free(output);
    free(resFromGPU);
    cudaErrorCheck(hipFree(inputGPU));
    cudaErrorCheck(hipFree(outputGPU));
    return EXIT_SUCCESS;
}