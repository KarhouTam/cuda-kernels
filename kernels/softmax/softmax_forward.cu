#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>

#include "common.h"

void softmax_cpu(float* input, float* output, const int M, const int N) {
    for (int m = 0; m < M; ++m) {
        float maxval = -INFINITY;
        const float* x = input + m * N;
        for (int n = 0; n < N; ++n) {
            maxval = maxval > x[n] ? maxval : x[n];
        }
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            s += exp(x[n] - maxval);
        }
        float* y = output + m * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

void online_softmax_cpu(float* input, float* output, const int M, const int N) {
    for (int m = 0; m < M; ++m) {
        const float* x = input + m * N;
        float maxval = -INFINITY;
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            if (maxval < x[n]) {
                s *= exp(maxval - x[n]);
                maxval = x[n];
            }
            s += exp(x[n] - maxval);
        }

        float* y = output + m * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

__global__ void softmax_kernel1(float* input, float* output, const int M,
                                const int N) {
    // naive implementation
    // one thread one row
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int idx = bid * blockDim.x + tid;
    if (idx < M) {
        float maxval = -INFINITY;
        const float* x = input + idx * N;
        for (int n = 0; n < N; ++n) {
            maxval = maxval > x[n] ? maxval : x[n];
        }
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            s += exp(x[n] - maxval);
        }
        float* const y = output + idx * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

__global__ void softmax_kernel2(float* input, float* output, const int M,
                                const int N) {
    // use more threads per row than kernel1
    // one warp (32 threads) process one row
    // use warp reduce functions
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    const int numWarps = warpsPerBlock * gridDim.x;
    const int idx = warpsPerBlock * blockIdx.x + warpId;
    for (int m = idx; m < M; m += numWarps) {
        // each lane (thread in a warp) calculate the maxval among
        // data with indices [landId, landId + 32, laneId + 64, ...]
        const float* x = input + m * N;
        float* const y = output + m * N;

        float maxval = -INFINITY;
        for (int i = laneId; i < N; i += warpSize) {
            maxval = fmaxf(maxval, x[i]);
        }
        // warp-reduce to calculate the MAX of maxval among all lanes
        // and the 0-th lane will store the result
        maxval = warpReduceMax(maxval);

        float sum = 0.0f;
        for (int i = laneId; i < N; i += warpSize) {
            sum += expf(x[i] - maxval);
        }

        sum = warpReduceSum(sum);
        for (int i = laneId; i < N; i += warpSize) {
            y[i] = expf(x[i] - maxval) / sum;
        }
    }
}

__global__ void online_softmax_kernel3(float* input, float* output, const int M,
                                       const int N) {
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    const int numWarps = warpsPerBlock * gridDim.x;
    const int idx = warpsPerBlock * blockIdx.x + warpId;
    for (int m = idx; m < M; m += numWarps) {
        const float* x = input + m * N;
        float* const y = output + m * N;
        float maxval = -INFINITY, sum = 0.0f, bigger;
        for (int i = laneId; i < N; i += warpSize) {
            bigger = fmaxf(maxval, x[i]);
            sum = sum * expf(maxval - bigger) + expf(x[i] - bigger);
        }

        float offsetMax, offsetSum;
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            __syncwarp();
            offsetMax = __shfl_xor_sync(0xFFFFFFFF, maxval, offset);
            offsetSum = __shfl_xor_sync(0xFFFFFFFF, sum, offset);
            if (offsetMax > maxval) {
                sum *= expf(maxval - offsetMax);
                maxval = offsetMax;
            } else {
                offsetSum *= expf(offsetMax - maxval);
            }
            sum += offsetSum;
        }
        for (int i = laneId; i < N; i += warpSize) {
            y[i] = expf(x[i] - maxval) / sum;
        }
    }
}

#define M 8196
#define N 8196
#define BLOCK_SIZE 128

int main(int argc, char** argv) {
    if (argc < 2) {
        fprintf(stderr, "Usage: softmax_forward <kernel> [blockSize]\n");
        return EXIT_FAILURE;
    }
    int kernel = atoi(argv[1]);

    int blockSize = BLOCK_SIZE;
    if (argc > 2) {
        blockSize = atoi(argv[2]);
    }

    float* input = (float*)malloc(M * N * sizeof(float));
    float* output = (float*)malloc(M * N * sizeof(float));
    float* resFromGPU = (float*)malloc(M * N * sizeof(float));
    initArrFloat(input, M * N);

    float *inputGPU, *outputGPU;
    cudaErrorCheck(hipMalloc(&inputGPU, M * N * sizeof(float)));
    cudaErrorCheck(hipMemcpy(inputGPU, input, M * N * sizeof(float),
                              hipMemcpyHostToDevice));
    cudaErrorCheck(hipMalloc(&outputGPU, M * N * sizeof(float)));

    float elapsedTime;
    online_softmax_cpu(input, output, M, N);

    switch (kernel) {
        case 1:
            benchmark_kernel(softmax_kernel1, M * N / blockSize, blockSize,
                             inputGPU, outputGPU, resFromGPU, M, N,
                             &elapsedTime);
            break;
        case 2:
            benchmark_kernel(softmax_kernel2, ceilDiv(M * 32, blockSize),
                             blockSize, inputGPU, outputGPU, resFromGPU, M, N,
                             &elapsedTime);
            break;
        case 3:
            benchmark_kernel(online_softmax_kernel3, ceilDiv(M * 32, blockSize),
                             blockSize, inputGPU, outputGPU, resFromGPU, M, N,
                             &elapsedTime);
            break;
        default:
            printf("Error: Invalid kernel type: %i\n", kernel);
            return EXIT_FAILURE;
    }

    if (checkResults(output, resFromGPU, M * N)) {
        printf(
            "softmax_forward kernel: %i | matrixSize: %i x %i | Times: %f ms | "
            "blockSize: %i\n",
            kernel, M, N, elapsedTime, blockSize);
    }
    free(input);
    free(output);
    free(resFromGPU);
    cudaErrorCheck(hipFree(inputGPU));
    cudaErrorCheck(hipFree(outputGPU));
    return EXIT_SUCCESS;
}