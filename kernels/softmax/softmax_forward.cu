#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hip/hip_runtime.h>

#include "common.h"

/* Softmax forward implementation

Usage: ./softmax_forward <kernel> [blockSize]
e.g. ./softmax_forward 1

softmax_forward_cpu(): CPU implementation

softmax_forward_kernel1(): Naive implementation on CUDA. Each thread handles
one row of the input.

softmax_forward_kernel2(): Optimized implementation on CUDA. Compares to
kernel1, each warp (32 threads) handles one row.

online_softmax_forward_kernel3(): Online softmax forward implementation on CUDA.
Also each warp handles one row of the input.

softmax_forward_kernel4(): Online softmax forward implementation on CUDA.
Using block reduce.

softmax_forward_kernel5(): Online softmax forward implementation on CUDA.
Each warp handles one row of the input.
Use float4 to acclerate memory access.

*/

void softmax_cpu(float* input, float* output, const int M, const int N) {
    for (int m = 0; m < M; ++m) {
        float maxval = -INFINITY;
        const float* x = input + m * N;
        for (int n = 0; n < N; ++n) {
            maxval = maxval > x[n] ? maxval : x[n];
        }
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            s += exp(x[n] - maxval);
        }
        float* y = output + m * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

void online_softmax_cpu(float* input, float* output, const int M, const int N) {
    for (int m = 0; m < M; ++m) {
        const float* x = input + m * N;
        float maxval = -INFINITY;
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            if (maxval < x[n]) {
                s *= exp(maxval - x[n]);
                maxval = x[n];
            }
            s += exp(x[n] - maxval);
        }

        float* y = output + m * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

__global__ void softmax_kernel1(float* input, float* output, const int M, const int N) {
    // naive implementation
    // one thread one row
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int idx = bid * blockDim.x + tid;
    if (idx < M) {
        float maxval = -INFINITY;
        const float* x = input + idx * N;
        for (int n = 0; n < N; ++n) {
            maxval = maxval > x[n] ? maxval : x[n];
        }
        float s = 0.0f;
        for (int n = 0; n < N; ++n) {
            s += exp(x[n] - maxval);
        }
        float* const y = output + idx * N;
        for (int n = 0; n < N; ++n) {
            y[n] = exp(x[n] - maxval) / s;
        }
    }
}

__global__ void softmax_kernel2(float* input, float* output, const int M, const int N) {
    // use more threads per row than kernel1
    // one warp (32 threads) process one row
    // use warp reduce functions
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    const int numWarps = warpsPerBlock * gridDim.x;
    const int idx = warpsPerBlock * blockIdx.x + warpId;
    for (int m = idx; m < M; m += numWarps) {
        // each lane (thread in a warp) calculate the maxval among
        // data with indices [landId, landId + 32, laneId + 64, ...]
        const float* x = input + m * N;
        float* const y = output + m * N;

        float maxval = -INFINITY;
        for (int i = laneId; i < N; i += warpSize) {
            maxval = fmaxf(maxval, x[i]);
        }
        // warp-reduce to calculate the MAX of maxval among all lanes
        // and the 0-th lane will store the result
        maxval = warpReduceMax(maxval);

        float sum = 0.0f;
        for (int i = laneId; i < N; i += warpSize) {
            sum += expf(x[i] - maxval);
        }

        sum = warpReduceSum(sum);
        for (int i = laneId; i < N; i += warpSize) {
            y[i] = expf(x[i] - maxval) / sum;
        }
    }
}

__global__ void online_softmax_kernel3(float* input, float* output, const int M, const int N) {
    //  one warp per row
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    const int numWarps = warpsPerBlock * gridDim.x;
    const int idx = warpsPerBlock * blockIdx.x + warpId;
    for (int m = idx; m < M; m += numWarps) {
        const float* x = input + m * N;
        float* const y = output + m * N;
        float maxval = -INFINITY, sum = 0.0f;
        for (int i = laneId; i < N; i += warpSize) {
            float xi = x[i];
            float newMax = fmaxf(maxval, xi);
            sum = sum * expf(maxval - newMax) + expf(xi - newMax);
            maxval = newMax;
        }

        float offsetMax, offsetSum;
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            offsetMax = __shfl_xor_sync(0xFFFFFFFF, maxval, offset);
            offsetSum = __shfl_xor_sync(0xFFFFFFFF, sum, offset);
            if (offsetMax > maxval) {
                sum *= expf(maxval - offsetMax);
                maxval = offsetMax;
            } else {
                offsetSum *= expf(offsetMax - maxval);
            }
            sum += offsetSum;
        }
        for (int i = laneId; i < N; i += warpSize) {
            y[i] = expf(x[i] - maxval) / sum;
        }
    }
}

__global__ void online_softmax_kernel4(float* input, float* output, const int M, const int N) {
    // one block per row
    extern __shared__ float shared[];
    const int laneId = threadIdx.x % warpSize;
    const int warpId = threadIdx.x / warpSize;
    const int warpsPerBlock = ceilDiv(blockDim.x, warpSize);
    const int dataPerWarp = ceilDiv(N, warpsPerBlock);
    const int start = dataPerWarp * warpId;
    const int end = min((warpId + 1) * dataPerWarp, N);
    const float* x = input + blockIdx.x * N;
    float* const y = output + blockIdx.x * N;

    float* const maxVals = shared;
    float* const sumVals = shared + warpsPerBlock;

    // Initialize maxval and sumval properly
    float maxval = -INFINITY, sumval = 0.f;

    // First pass: compute max and sum for this warp's data range
    for (int i = start + laneId; i < end; i += warpSize) {
        float xi = x[i];
        float newMax = fmaxf(maxval, xi);
        sumval = sumval * expf(maxval - newMax) + expf(xi - newMax);
        maxval = newMax;
    }

    // Warp reduction to get warp-level max and sum
    float warpMaxval = warpReduceMax(maxval);
    sumval *= expf(maxval - warpMaxval);
    float warpSumval = warpReduceSum(sumval);

    // Store warp results to shared memory
    if (laneId == 0) {
        maxVals[warpId] = warpMaxval;
        sumVals[warpId] = warpSumval;
    }
    __syncthreads();

    // Block reduction using warp 0
    if (warpId == 0) {
        maxval = (laneId < warpsPerBlock) ? maxVals[laneId] : -INFINITY;
        sumval = (laneId < warpsPerBlock) ? sumVals[laneId] : 0.0f;

        // Reduce across warps
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            float otherMax = __shfl_xor_sync(0xFFFFFFFF, maxval, offset);
            float otherSum = __shfl_xor_sync(0xFFFFFFFF, sumval, offset);

            if (maxval < otherMax) {
                sumval *= expf(maxval - otherMax);
                maxval = otherMax;
            } else if (maxval > otherMax) {
                otherSum *= expf(otherMax - maxval);
            }
            sumval += otherSum;
        }

        // First thread writes final results
        if (laneId == 0) {
            maxVals[0] = maxval;
            sumVals[0] = sumval;
        }
    }
    __syncthreads();

    // Final computation using block-wide max and sum
    float blockMax = maxVals[0];
    float blockSum = sumVals[0];

    // Write final results
    for (int i = start + laneId; i < end; i += warpSize) {
        y[i] = expf(x[i] - blockMax) / blockSum;
    }
}
__global__ void online_softmax_kernel5(float* __restrict__ input, float* __restrict__ output, const int M,
                                       const int N) {
    // this kernel is f*cking faster than any other kernels!
    // use float4 to acclerate memory access
    // each warp (32 threads) handles one row
    // TODO: fix bug of misaligned data memory access
    using f128 = Package128<float>;
    const int tid = threadIdx.x;
    const int warpId = tid / warpSize;
    const int laneId = tid % warpSize;
    const int warpsPerBlock = blockDim.x / warpSize;
    int row = warpsPerBlock * blockIdx.x + warpId;
    if (row < M) {
        float* x = input + row * N;
        float* y = output + row * N;
        float laneMax = -INFINITY, laneSum = 0.0f;
        int i = ceilDiv(N, f128::size) + laneId - warpSize;
        while ((i + 1) * f128::size >= N) {
            for (int k = 0; k < f128::size; ++k) {
                if (i * f128::size + k >= N) {
                    break;
                }
                float newLaneMax = fmaxf(laneMax, x[i * f128::size + k]);
                laneSum = laneSum * expf(laneMax - newLaneMax) + expf(x[i * f128::size + k] - newLaneMax);
                laneMax = newLaneMax;
            }
            i -= warpSize;
        }

        for (; i >= 0; i -= warpSize) {
            f128 xi = load128cs(x + i * f128::size);
            float packMax = -INFINITY, packSum = 0.0f;
#pragma unroll
            for (int k = 0; k < f128::size; ++k) {
                float newPackMax = fmaxf(packMax, xi[k]);
                packSum = expf(packMax - newPackMax) * packSum + expf(xi[k] - newPackMax);
                packMax = newPackMax;
            }
            float newLaneMax = fmaxf(laneMax, packMax);
            laneSum = laneSum * expf(laneMax - newLaneMax) + packSum * expf(packMax - newLaneMax);
            laneMax = newLaneMax;
        }

        float maxVal = laneMax, sumVal = laneSum;
        for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
            float offsetMax = __shfl_xor_sync(0xFFFFFFFF, maxVal, offset);
            float offsetSum = __shfl_xor_sync(0xFFFFFFFF, sumVal, offset);
            if (maxVal > offsetMax) {
                sumVal += expf(offsetMax - maxVal) * offsetSum;
            } else {
                sumVal = sumVal * expf(maxVal - offsetMax) + offsetSum;
                maxVal = offsetMax;
            }
        }

        i = ceilDiv(N, f128::size) + laneId - warpSize;
        while ((i + 1) * f128::size >= N) {
            for (int k = 0; k < f128::size; ++k) {
                if (i * f128::size + k >= N) {
                    break;
                }
                y[i * f128::size + k] = expf(x[i * f128::size + k] - maxVal) / sumVal;
            }
            i -= warpSize;
        }

        for (; i >= 0; i -= warpSize) {
            f128 out;
            f128 xi = load128cs(x + i * f128::size);
#pragma unroll
            for (int k = 0; k < f128::size; ++k) {
                out[k] = expf(xi[k] - maxVal) / sumVal;
            }
            store128cs(y + i * f128::size, out);
        }
    }
}
#define M 8196
#define N 8196
#define BLOCK_SIZE 128
#define REPEAT_TIMES 100

int main(int argc, char** argv) {
    if (argc < 2) {
        fprintf(stderr, "Usage: softmax_forward <kernel> [blockSize] [benchmarkRepeatTimes]\n");
        return EXIT_FAILURE;
    }
    int kernel = atoi(argv[1]);

    int blockSize = BLOCK_SIZE;
    if (argc > 2) {
        blockSize = atoi(argv[2]);
    }
    int repeatTimes = REPEAT_TIMES;
    if (argc > 3) {
        repeatTimes = atoi(argv[3]);
    }

    float* input = (float*)malloc(M * N * sizeof(float));
    float* output = (float*)malloc(M * N * sizeof(float));
    float* resFromGPU = (float*)malloc(M * N * sizeof(float));
    initArrFloat(input, M * N);

    float *inputGPU, *outputGPU;
    cudaErrorCheck(hipMalloc(&inputGPU, M * N * sizeof(float)));
    cudaErrorCheck(hipMemcpy(inputGPU, input, M * N * sizeof(float), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMalloc(&outputGPU, M * N * sizeof(float)));

    online_softmax_cpu(input, output, M, N);

    switch (kernel) {
        case 1:
            softmax_kernel1<<<ceilDiv(M, blockSize), blockSize>>>(inputGPU, outputGPU, M, N);
            break;
        case 2:
            softmax_kernel2<<<ceilDiv(M * 32, blockSize), blockSize>>>(inputGPU, outputGPU, M, N);
            break;

        case 3:
            online_softmax_kernel3<<<ceilDiv(M * 32, blockSize), blockSize>>>(inputGPU, outputGPU, M, N);
            break;

        case 4:
            online_softmax_kernel4<<<M, blockSize, blockSize / 32 * 2 * sizeof(float)>>>(inputGPU,
                                                                                         outputGPU, M, N);
            break;

        case 5:
            online_softmax_kernel5<<<ceilDiv(M * 32, blockSize), blockSize, 0>>>(inputGPU, outputGPU, M, N);
            break;
        default:
            printf("Error: Invalid kernel type: %i\n", kernel);
            return EXIT_FAILURE;
    }
    cudaErrorCheck(hipDeviceSynchronize());
    cudaErrorCheck(hipMemcpy(resFromGPU, outputGPU, M * N * sizeof(float), hipMemcpyDeviceToHost));

    float elapsedTime = 0.0f;
    if (checkResults(output, resFromGPU, M * N)) {
        switch (kernel) {
            case 1:
                benchmarkKernel(repeatTimes, softmax_kernel1, M * N / blockSize, blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 2:
                benchmarkKernel(repeatTimes, softmax_kernel2, ceilDiv(M * 32, blockSize), blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 3:
                benchmarkKernel(repeatTimes, online_softmax_kernel3, ceilDiv(M * 32, blockSize), blockSize,
                                0, 0, &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 4:
                benchmarkKernel(repeatTimes, online_softmax_kernel4, M, blockSize,
                                blockSize / 32 * 2 * sizeof(float), 0, &elapsedTime, inputGPU, outputGPU, M,
                                N);
                break;
            case 5:
                benchmarkKernel(repeatTimes, online_softmax_kernel5, ceilDiv(M * 32, blockSize), blockSize,
                                0, 0, &elapsedTime, inputGPU, outputGPU, M, N);
                break;
        }
        printf(
            "softmax_forward kernel: %i | matrixSize: %i x %i | Times: %f ms | "
            "blockSize: %i\n",
            kernel, M, N, elapsedTime, blockSize);
    }

    free(input);
    free(output);
    free(resFromGPU);
    cudaErrorCheck(hipFree(inputGPU));
    cudaErrorCheck(hipFree(outputGPU));
    return EXIT_SUCCESS;
}