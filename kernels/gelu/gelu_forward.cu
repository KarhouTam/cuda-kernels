#include "hip/hip_runtime.h"
#include "common.h"

/* ReLU forward implementation

Usage: ./gelu_forward <kernel> [blockSize]
e.g. ./gelu_forward 1

gelu_forward_cpu(): CPU implementation

gelu_forward_kernel1(): Naive implementation on CUDA. Each thread handles
one row of the input.

gelu_forward_kernel2(): Optimized implementation on CUDA. Compares to
kernel1, each warp (32 threads) handles one row.

gelu_forward_kernel3(): Optimized implementation on CUDA. Compares to
kernel2, using float4.

gelu_forward_kernel4(): Optimized implementation on CUDA. Each thread handles one FLOAT4.
*/
#define GELU_SCALING_FACTOR sqrtf(2.0f / M_PI)
void gelu_cpu(float* input, float* output, const int M, const int N) {
    for (int m = 0; m < M; ++m) {
        const float* x = input + m * N;
        float* const y = output + m * N;
        for (int n = 0; n < N; ++n) {
            float xn = x[n];
            float cube = 0.044715f * xn * xn * xn;
            y[n] = 0.5f * xn * (1.0f + tanhf(GELU_SCALING_FACTOR * (xn + cube)));
        }
    }
}

__global__ void gelu_forward_kernel1(const float* input, float* output, const int M, const int N) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < M) {
        const float* x = input + idx * N;
        float* const y = output + idx * N;
        for (int n = 0; n < N; ++n) {
            float xn = x[n];
            float cube = 0.044715f * xn * xn * xn;
            y[n] = 0.5f * xn * (1.0f + tanhf(GELU_SCALING_FACTOR * (xn + cube)));
        }
    }
}

__global__ void gelu_forward_kernel2(float* input, float* output, int M, int N) {
    // each warp handles one row of the input
    int warpsPerBlock = blockDim.x / warpSize;
    int warpId = threadIdx.x / warpSize;
    int laneId = threadIdx.x % warpSize;
    int numWarps = gridDim.x * warpsPerBlock;
    for (int row = blockIdx.x * warpsPerBlock + warpId; row < M; row += numWarps)
        if (row < M) {
            float* const x = input + row * N;
            float* const y = output + row * N;

            for (int i = laneId; i < N; i += warpSize) {
                float xi = x[i];
                float cube = 0.044715f * xi * xi * xi;
                y[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
            }
        }
}

__global__ void gelu_forward_kernel3(float* input, float* output, int M, int N) {
    // each warp handles one row of the input
    // use floar4 to acclerate memory accessing
    // but seems improvement is not significant
    using f128 = Package128<float>;
    int warpsPerBlock = blockDim.x / warpSize;
    int warpId = threadIdx.x / warpSize;
    int laneId = threadIdx.x % warpSize;
    int numWarps = gridDim.x * warpsPerBlock;
    for (int row = blockIdx.x * warpsPerBlock + warpId; row < M; row += numWarps)
        if (row < M) {
            float* const x = input + row * N;
            float* const y = output + row * N;

            for (int i = laneId * f128::size; i < N; i += warpSize * f128::size) {
                f128 packedX = load128(x + i);
                f128 out;
#pragma unroll
                for (int k = 0; k < f128::size; ++k) {
                    float xik = packedX[k];
                    float cube = 0.044715f * xik * xik * xik;
                    out[k] = 0.5f * xik * (1.0f + tanhf(GELU_SCALING_FACTOR * (xik + cube)));
                }
                store128(y + i, out);
            }
        }
}

__global__ void gelu_forward_kernel4(float* input, float* output, int M, int N) {
    using f128 = Package128<float>;
    const int idx = (blockIdx.x * blockDim.x + threadIdx.x) * f128::size;
    if (idx + f128::size < M * N) {
        f128 packedX = load128cs(input + idx);
        f128 packedY;
        for (int k = 0; k < f128::size; ++k) {
            float xik = packedX[k];
            float cube = 0.044715f * xik * xik * xik;
            packedY[k] = 0.5f * xik * (1.0f + tanhf(GELU_SCALING_FACTOR * (xik + cube)));
        }
        store128(output + idx, packedY);
    } else {
        for (int i = idx; i < M * N; ++i) {
            float xi = input[i];
            float cube = 0.044715f * xi * xi * xi;
            output[i] = 0.5f * xi * (1.0f + tanhf(GELU_SCALING_FACTOR * (xi + cube)));
        }
    }
}

#define M 8192
#define N 8192
#define BLOCK_SIZE 128
#define REPEAT_TIMES 100

int main(int argc, char** argv) {
    if (argc < 2) {
        fprintf(stderr, "Usage: gelu_forward <kernel> [blockSize] [benchmarkRepeatTimes]\n");
        return EXIT_FAILURE;
    }
    int kernel = atoi(argv[1]);

    int blockSize = BLOCK_SIZE;
    if (argc > 2) {
        blockSize = atoi(argv[2]);
    }
    int repeatTimes = REPEAT_TIMES;
    if (argc > 3) {
        repeatTimes = atoi(argv[3]);
    }

    float* input = (float*)malloc(M * N * sizeof(float));
    float* output = (float*)malloc(M * N * sizeof(float));
    float* resFromGPU = (float*)malloc(M * N * sizeof(float));
    initArrFloat(input, M * N);

    float *inputGPU, *outputGPU;
    cudaErrorCheck(hipMalloc(&inputGPU, M * N * sizeof(float)));
    cudaErrorCheck(hipMemcpy(inputGPU, input, M * N * sizeof(float), hipMemcpyHostToDevice));
    cudaErrorCheck(hipMalloc(&outputGPU, M * N * sizeof(float)));

    float elapsedTime = 0.0f;

    gelu_cpu(input, output, M, N);

    switch (kernel) {
        case 1:
            gelu_forward_kernel1<<<M * N / blockSize, blockSize>>>(inputGPU, outputGPU, M, N);
            break;
        case 2:
            gelu_forward_kernel2<<<M * N / blockSize, blockSize>>>(inputGPU, outputGPU, M, N);
            break;
        case 3:
            gelu_forward_kernel3<<<M * N / blockSize, blockSize>>>(inputGPU, outputGPU, M, N);
            break;
        case 4:
            gelu_forward_kernel4<<<ceilDiv(M * N, (blockSize * Package128<float>::size)), blockSize>>>(
                inputGPU, outputGPU, M, N);
            break;
        default:
            printf("Error: Invalid kernel type: %i\n", kernel);
            return EXIT_FAILURE;
    }
    cudaErrorCheck(hipMemcpy(resFromGPU, outputGPU, M * N * sizeof(float), hipMemcpyDeviceToHost));
    cudaErrorCheck(hipDeviceSynchronize());

    if (checkResults(output, resFromGPU, M * N)) {
        switch (kernel) {
            case 1:
                benchmarkKernel(repeatTimes, gelu_forward_kernel1, M * N / blockSize, blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 2:
                benchmarkKernel(repeatTimes, gelu_forward_kernel2, M * N / blockSize, blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 3:
                benchmarkKernel(repeatTimes, gelu_forward_kernel3, M * N / blockSize, blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
            case 4:
                benchmarkKernel(repeatTimes, gelu_forward_kernel4,
                                ceilDiv(M * N, (blockSize * Package128<float>::size)), blockSize, 0, 0,
                                &elapsedTime, inputGPU, outputGPU, M, N);
                break;
        }
        printf(
            "softmax_forward kernel: %i | matrixSize: %i x %i | Times: "
            "%f ms | "
            "blockSize: %i\n",
            kernel, M, N, elapsedTime, blockSize);
    }
    free(input);
    free(output);
    free(resFromGPU);
    cudaErrorCheck(hipFree(inputGPU));
    cudaErrorCheck(hipFree(outputGPU));
    return EXIT_SUCCESS;
}