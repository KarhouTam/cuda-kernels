#include "hip/hip_runtime.h"
#include "common.h"

void gemm_cpu(const float *A, const float *B, const float *C, float *const D,
              const int M, const int N, const int K) {
    // D = A * B + C
    // A: M x K
    // B: K x N
    // C: M x N
    // D: M x N
    for (int m = 0; m < M; ++m) {
        for (int n = 0; n < N; ++n) {
            float val = 0.f;
            for (int k = 0; k < K; ++k) {
                val += A[m * K + k] * B[k * N + n];
            }
            D[m * N + n] = val + C[m * N + n];
        }
    }
}

__global__ void gemm_kernel1(const float *A, const float *B, const float *C,
                             float *const D, const int M, const int N,
                             const int K) {
    // naive implementation
    // each thread calculates one row of D (M rows in total, one row has N
    // elements)
    const int m = blockDim.x * blockIdx.x + threadIdx.x;
    for (int n = 0; n < N; ++n) {
        float val = 0.f;
        for (int k = 0; k < K; ++k) {
            val += A[m * K + k] * B[k * N + n];
        }
        D[m * N + n] = val + C[m * N + n];
    }
}
template <int blockSize>
__global__ void gemm_kernel2(const float *A, const float *B, const float *C,
                             float *const D, const int M, const int N,
                             const int K) {
    __shared__ float sharedA[blockSize][blockSize];
    __shared__ float sharedB[blockSize][blockSize];
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float val = 0;
    if (row < M && col < N) {
        val = C[row * N + col];
    }

    for (int k = 0; k < K; k += blockSize) {
        if (row < M && (k + threadIdx.x) < K) {
            sharedA[threadIdx.y][threadIdx.x] = A[row * K + k + threadIdx.x];
        } else {
            sharedA[threadIdx.y][threadIdx.x] = 0.0f;
        }

        if ((k + threadIdx.y) < K && col < N) {
            sharedB[threadIdx.y][threadIdx.x] = B[(k + threadIdx.y) * N + col];
        } else {
            sharedB[threadIdx.y][threadIdx.x] = 0.0f;
        }

        __syncthreads();

        if (row < M && col < N) {
            for (int i = 0; i < blockSize; ++i) {
                val += sharedA[threadIdx.y][i] * sharedB[i][threadIdx.x];
            }
        }

        __syncthreads();
    }

    if (row < M && col < N) {
        D[row * N + col] = val;
    }
}

constexpr unsigned int M = 2048;
constexpr unsigned int K = 512;
constexpr unsigned int N = 2048;
constexpr unsigned int BLOCK_SIZE = 256;
constexpr unsigned int BLOCK_SIZE_KERNEL2 = 16;

int main(int argc, char **argv) {
    if (argc < 2) {
        fprintf(stderr, "Usage: gemm_forward <kernel> [blockSize]\n");
        return EXIT_FAILURE;
    }
    int kernel = atoi(argv[1]);

    unsigned int blockSize = BLOCK_SIZE;
    if (argc > 2) {
        blockSize = atoi(argv[2]);
    }

    float *A = (float *)malloc(M * K * sizeof(float));
    float *B = (float *)malloc(K * N * sizeof(float));
    float *C = (float *)malloc(M * N * sizeof(float));
    float *D = (float *)malloc(M * N * sizeof(float));
    float *resFromGPU = (float *)malloc(M * N * sizeof(float));
    initArrFloat(A, M * K);
    initArrFloat(B, K * N);
    initArrFloat(C, M * N);

    float *AGPU, *BGPU, *CGPU, *DGPU;

    cudaErrorCheck(hipMalloc(&AGPU, M * K * sizeof(float)));
    cudaErrorCheck(
        hipMemcpy(AGPU, A, M * K * sizeof(float), hipMemcpyHostToDevice));

    cudaErrorCheck(hipMalloc(&BGPU, K * N * sizeof(float)));
    cudaErrorCheck(
        hipMemcpy(BGPU, B, K * N * sizeof(float), hipMemcpyHostToDevice));

    cudaErrorCheck(hipMalloc(&CGPU, M * N * sizeof(float)));
    cudaErrorCheck(
        hipMemcpy(CGPU, C, M * N * sizeof(float), hipMemcpyHostToDevice));

    cudaErrorCheck(hipMalloc(&DGPU, M * N * sizeof(float)));

    float elapsedTime;
    gemm_cpu(A, B, C, D, M, N, K);

    switch (kernel) {
        case 1:
            gemm_kernel1<<<ceilDiv(M, blockSize), blockSize>>>(AGPU, BGPU, CGPU,
                                                               DGPU, M, N, K);
            break;
        case 2: {
            blockSize = BLOCK_SIZE_KERNEL2 * BLOCK_SIZE_KERNEL2;
            dim3 blockDim(BLOCK_SIZE_KERNEL2, BLOCK_SIZE_KERNEL2);
            dim3 gridDim(ceilDiv(N, BLOCK_SIZE_KERNEL2),
                         ceilDiv(M, BLOCK_SIZE_KERNEL2));
            gemm_kernel2<BLOCK_SIZE_KERNEL2>
                <<<gridDim, blockDim>>>(AGPU, BGPU, CGPU, DGPU, M, N, K);
            break;
        }
        default:
            printf("Error: Invalid kernel type: %i\n", kernel);
            return EXIT_FAILURE;
    }
    cudaErrorCheck(hipDeviceSynchronize());
    cudaErrorCheck(hipMemcpy(resFromGPU, DGPU, M * N * sizeof(float),
                              hipMemcpyDeviceToHost));
    if (checkResults(D, resFromGPU, M * N)) {
        switch (kernel) {
            case 1:
                benchmarkKernel(gemm_kernel1, ceilDiv(M, blockSize), blockSize,
                                &elapsedTime, AGPU, BGPU, CGPU, DGPU, M, N, K);
                break;
            case 2:
                benchmarkKernel(gemm_kernel2<BLOCK_SIZE_KERNEL2>,
                                dim3(ceilDiv(N, BLOCK_SIZE_KERNEL2),
                                     ceilDiv(M, BLOCK_SIZE_KERNEL2)),
                                dim3(BLOCK_SIZE_KERNEL2, BLOCK_SIZE_KERNEL2),
                                &elapsedTime, AGPU, BGPU, CGPU, DGPU, M, N, K);
                break;
            default:
                printf("Error: Invalid kernel type: %i\n", kernel);
                return EXIT_FAILURE;
        }
        printf(
            "gemm_forward kernel: %i | A: (%i, %i), B: (%i, %i), C: (%i, "
            "%i) | "
            "Times: %f ms | "
            "blockSize: %i\n",
            kernel, M, K, K, N, M, N, elapsedTime, blockSize);
    }

    free(A);
    free(B);
    free(C);
    free(D);
    free(resFromGPU);
    cudaErrorCheck(hipFree(AGPU));
    cudaErrorCheck(hipFree(BGPU));
    cudaErrorCheck(hipFree(CGPU));
    cudaErrorCheck(hipFree(DGPU));
    return EXIT_SUCCESS;
}